
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 128
#define SIZE 10
#define BUCKETS 2

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ void hist(unsigned char*array, long size, unsigned int*histo){
	int i = blockIdx.x *blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	
	if(i>=size){
		return;
	}
	while (i<size){
		int value = array[i];
		int bin = (value % BUCKETS);
		atomicAdd(&(histo[bin]),1);
		i+=stride;
	}
}

int main(){
	unsigned char * array =(unsigned char*)malloc(sizeof(char)*SIZE);
	unsigned int* histo = (unsigned int*)malloc(sizeof(int)*SIZE);

	for(int i=0; i<SIZE; i++){
		array[i]=i;
	}
	printf("\n");

	unsigned char*dArray;
	hipMalloc(&dArray, SIZE);
	hipMemcpy(dArray,array,SIZE,hipMemcpyHostToDevice);

	unsigned int * dHist;
	hipMalloc(&dHist,BUCKETS*sizeof(int));
	hipMemset(dHist,0,BUCKETS*sizeof(int));

	dim3 block(32);
	dim3 grid((SIZE+block.x-1)/block.x);

	hipDeviceSynchronize();
	double t0 = get_clock();
	hist<<<grid,block>>>(dArray,SIZE,dHist);
	hipDeviceSynchronize();
	double t1 = get_clock();

	hipMemcpy(histo,dHist,BUCKETS *sizeof(int),hipMemcpyDeviceToHost);

	for (int i=0;i<BUCKETS; i++){
		printf(" %d |", histo[i]);
	}
	printf("\n");
	printf("Time: %f ns\n", (1000000000.0*(t1-t0)));
	printf("\n");
}
