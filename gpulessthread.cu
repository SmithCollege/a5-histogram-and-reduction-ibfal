
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 128
#define SIZE 25000

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


__global__ 
void reduction(int* input, int*out){//sum reduction
	__shared__ float partial[2*BLOCK_SIZE];
	
	unsigned int t = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;
	partial[t]=input[start+t];
	partial[blockDim.x + t]=input[start + blockDim.x + t];
	
	for(unsigned int stride =blockDim.x; stride>=1; stride>>=1){
		__syncthreads();
		if(t<stride){
			partial[t]+= partial[t+stride];
			out[0]= partial[t];
		}
	}	
} 

int main (){
	int* values, *dv, *sum, *ds;
	values = (int*)malloc(SIZE*sizeof(int));
	sum = (int*)malloc(1*sizeof(int));
	hipMallocManaged(&dv, SIZE*sizeof(int));
	hipMallocManaged(&ds, 1*sizeof(int));

	for (int i = 0; i < SIZE; i++) {
		values[i] = rand()%(10-0+1);
	}
	
	//gpu sum
	hipMemcpy(dv, values, SIZE*sizeof(int), hipMemcpyHostToDevice); 
	double t0 = get_clock();
	reduction<<<1, BLOCK_SIZE>>>(dv,ds);
	hipDeviceSynchronize();
	double t1 = get_clock();
	
	hipMemcpy(sum, ds, 1*sizeof(int), hipMemcpyDeviceToHost);
	//need kernel here
	printf("%d\n", sum[0]);	

	printf("\n");
	printf("Time: %f ns\n", (1000000000.0*(t1-t0)));
}
