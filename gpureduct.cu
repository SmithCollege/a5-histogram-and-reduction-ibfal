
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 10
#define BLOCK_SIZE 32

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ 
void reduction(int* input, int*out){//sum reduction
	__shared__ float partial[2*BLOCK_SIZE];
	
	unsigned int t = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;
	partial[t]=input[start+t];
	partial[blockDim.x + t]=input[start + blockDim.x + t];
	
	for(unsigned int stride =1; stride<= blockDim.x; stride*=2){
		__syncthreads();
		if(t%stride ==0){
			partial[2*t]+= partial[2*t+stride];
			out[0]= partial[2*t];
		}
	}	
}

int main (){
	int N = 15;
	int* values, *dv, *sum, *ds;
	values = (int*)malloc(N*sizeof(int));
	sum = (int*)malloc(1*sizeof(int));
	hipMallocManaged(&dv, N*sizeof(int));
	hipMallocManaged(&ds, 1*sizeof(int));

	for (int i = 0; i < SIZE; i++) {
		values[i] = rand()%(N-0+1);
	}
	//to print array
	//for (int i = 0; i < SIZE; i++) {
		//printf("%d ", values[i]);
		//}

	//printf("\n");
	
	//cpu sum
	int s = 0;
	for (int i = 0; i < SIZE; i++) {
		s += values[i];
	}
	printf("%d\n ", s);
	printf("\n");
	
	//gpu sum
	hipMemcpy(dv, values, N*sizeof(int), hipMemcpyHostToDevice); 
	double t0 = get_clock();
	reduction<<<1, BLOCK_SIZE>>>(dv,ds);
	hipDeviceSynchronize();
	double t1 = get_clock();
	
	hipMemcpy(sum, ds, 1*sizeof(int), hipMemcpyDeviceToHost);
	//need kernel here
	printf("%d\n", sum[0]);	

	printf("\n");
	printf("Time: %f ns\n", (1000000000.0*(t1-t0)));
}
