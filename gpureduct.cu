#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 10
#define BLOCK_SIZE 32

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ 
void reduction(int* input. int*out){//sum reduction
	__shared__ float partial[2*BLOCK_SIZE];
	
	unsigned int t = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;
	partial[t]=input[start+t];
	partial[blockDim+t]=input[start+blockDim.x+t];
	
	for(unsigned int stride =1; stride<= blockDim.x; stride*=2){
		__syncthreads();
		if(t%stride ==0){
			partial[2*t]+= partial[2*t+stride];
		}
	}
	out[0]=	partial[2*t];
}

int sum(int * arr){
	int sum = 0;
	for (int i = 0; i < SIZE; i++) {
		sum += arr[i];
	}
	return sum;
}

int main (){
	int N = 15;
	int* values, *sum;
	hipMallocManaged(&values, sizeof(int) * BLOCK_SIZE);
	hipMallocManaged(&sum, sizeof(int));

	for (int i = 0; i < SIZE; i++) {
		values[i] = rand()%(N-0+1);
	}

	for (int i = 0; i < SIZE; i++) {
		printf("%d ", values[i]);
		}

	printf("\n");
	//cpu sum
	printf("%d\n ", sum(values));
	
	printf("\n");
	//gpu sum
	//need kernel here
	printf("%d\n", sum[0]);	

}
